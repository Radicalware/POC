#include "hip/hip_runtime.h"
#include "GPUCore.cuh"
#include "Timer.h"
#include "OS.h"
#include "CudaBridge.cuh"


__global__ void GPU::ParseResultColumnIdx(
    ColumnSummary* FvSummaries, 
    RA::StatsGPU* FvStats, const ColumnData* FvColumnData, 
    const xint FnColumnCount, const xint FnRowCount,
    const xint FnReloop)
{
    auto Col = RA::Device::GetThreadID();
    if (Col >= FnColumnCount)
    {
        //RA::Device::Print(blockIdx, threadIdx);
        return;
    }
    
    const auto& LvColumnValues = FvColumnData[Col];
    auto& LoStats = FvStats[Col];
    auto& LoSummary = FvSummaries[Col];

    for (xint l = 0; l < FnReloop; l++)
    {
        for (xint i = 0; i < FnRowCount; i++)
            LoStats << LvColumnValues.MvDeviceRows[i];
    }
    
    LoSummary.SetGPU(FnRowCount, LoStats);
}

GPU::Core::Core(const xstring& FsFilePath): APU::Core(FsFilePath)
{
}

GPU::Core::~Core()
{
    HostDelete(MoHost.MvColumnData);
}

void GPU::Core::ConfigureColumnValues()
{
    Begin();
    APU::Core::ConfigureColumnValues();

    MoHost.MvColumnData = new ColumnData[GetColumnCount()];
    for (xint Col = 0; Col < GetColumnCount(); Col++)
        MoHost.MvColumnData[Col].Initialize(MoHost.MvColumnValues[Col]);

    MoDevice.MvColumnData = 
        RA::Host::AllocateArrOnDevice<ColumnData>(
        MoHost.MvColumnData, RA::Allocate(GetColumnCount(), 
            sizeof(ColumnData)));

    Rescue();
}

void GPU::Core::ParseResults(const bool FbForceRestart)
{
    Begin();
    if(!FbForceRestart)
        if (MbParsed && MoDevice.MoResultStats.Size())
            return;

    const auto LmStatOps = xmap<RA::EStatOpt, xint>{
        {RA::EStatOpt::AVG, 0},{RA::EStatOpt::STOCH, 0},{RA::EStatOpt::SD, 0}
    };

    MoHost.MvStatsGPU = MKP<RA::StatsGPU[]>(GetColumnCount());
    for (auto& LoStat : MoHost.MvStatsGPU)
        LoStat.Construct(0, LmStatOps);

    MoDevice.MoResultStats = RA::CudaBridge<RA::StatsGPU>(MoHost.MvStatsGPU, MoHost.MvStatsGPU.GetLength());
    MoDevice.MoResultStats.AllocateDevice();
    MoDevice.MoResultStats.CopyHostToDeviceAsync();
    MoDevice.MoResultStats.SyncStream();

    cout << "Column Count: " << RA::FormatNum(GetColumnCount()) << endl;
    const auto [LvGrid, LvBlock] = RA::Host::GetDimensions3D(GetColumnCount());
    
    SetCudaMaxMem(GPU::ParseResultColumnIdx);
    RA::Host::PrintGridBlockDims(LvGrid, LvBlock);
    MoDevice.MoColumnSummaries = RA::CudaBridge<ColumnSummary>::ARRAY::RunGPU(
        RA::Allocate(GetColumnCount(), sizeof(ColumnSummary)),
        LvGrid, LvBlock,
        &GPU::ParseResultColumnIdx,
        MoDevice.MoResultStats.GetDevice(),
        MoDevice.MvColumnData, GetColumnCount(), GetRowCount(),
        SnReloop
    );
    MoDevice.MoColumnSummaries.SyncStream();
    MoDevice.MoColumnSummaries.CopyDeviceToHost();
    MoDevice.MoColumnSummaries.SyncAll();

    MoHost.MvSummaries = MoDevice.MoColumnSummaries.GetShared();

    Rescue();
}
