#include "hip/hip_runtime.h"
#include "GPUCore.cuh"
#include "Timer.h"
#include "OS.h"
#include "CudaBridge.cuh"

#include <cmath>

__global__ void GPU::ParseResultColumnIdx(
    ColumnSummary* FvSummaries, 
    RA::StatsGPU* FvStats, const ColumnData* FvColumnData, 
    const xint FnColumnCount, const xint FnRowCount)
{
    auto Col = RA::Device::GetThreadID();
    if (Col >= FnColumnCount)
        return;
    
    const auto& LvColumnValues = FvColumnData[Col];
    auto& LoStats = FvStats[Col];
    auto& LoSummary = FvSummaries[Col];

    for (xint i = 0; i < FnRowCount; i++)
        LoStats << LvColumnValues.MvDeviceRows[i];
    
    LoSummary.SetGPU(FnRowCount, LoStats);
}

GPU::Core::Core(const xstring& FsFilePath): APU::Core(FsFilePath)
{
    const auto [LnGrid, LnBlock] = GetGridBlockConfig(20000);
}

GPU::Core::~Core()
{
    HostDelete(MoHost.MvColumnData);
}

void GPU::Core::ConfigureColumnValues()
{
    Begin();
    APU::Core::ConfigureColumnValues();

    MoHost.MvColumnData = new ColumnData[GetColumnCount()];
    for (xint Col = 0; Col < GetColumnCount(); Col++)
        MoHost.MvColumnData[Col].Initialize(MoHost.MvColumnValues[Col]);

    MoDevice.MvColumnData = 
        RA::Host::AllocateArrOnDevice<ColumnData>(
        MoHost.MvColumnData, RA::Allocate(GetColumnCount(), 
            sizeof(ColumnData)));

    Rescue();
}

void GPU::Core::ParseResults(const bool FbForceRestart)
{
    Begin();
    if(!FbForceRestart)
        if (MbParsed && MoDevice.MoResultStats.Size())
            return;

    const auto LnRetAllocate = RA::Allocate(GetColumnCount(), sizeof(ColumnData));
    const auto LmStatOps = xmap<RA::EStatOpt, xint>{
        {RA::EStatOpt::AVG, 0},{RA::EStatOpt::STOCH, 0},{RA::EStatOpt::SD, 0}
    };

    MoHost.MvStatsGPU = MKP<RA::StatsGPU[]>(GetColumnCount());
    for (auto& LoStat : MoHost.MvStatsGPU)
        LoStat.Construct(0, LmStatOps);

    MoDevice.MoResultStats = RA::CudaBridge<RA::StatsGPU>(MoHost.MvStatsGPU, MoHost.MvStatsGPU.GetLength());
    MoDevice.MoResultStats.AllocateHost();
    MoDevice.MoResultStats.AllocateDevice();
    MoDevice.MoResultStats.CopyHostToDeviceAsync();
    MoDevice.MoResultStats.SyncStream();

    //const auto [LnGrid, LnBlock] = RA::Host::GetDimensions3D(GetColumnCount());
    cout << "Column Count: " << GetColumnCount() << endl;
    const auto [LnGrid, LnBlock] = GetGridBlockConfig();
    

    MoDevice.MoColumnSummaries = RA::CudaBridge<ColumnSummary>::ARRAY::RunGPU(
        RA::Allocate(GetColumnCount(), sizeof(ColumnSummary)),
        LnGrid, LnBlock,
        &GPU::ParseResultColumnIdx,
        MoDevice.MoResultStats.GetDevice(),
        MoDevice.MvColumnData, GetColumnCount(), GetRowCount()
    );
    MoDevice.MoColumnSummaries.SyncStream();
    MoDevice.MoColumnSummaries.CopyDeviceToHost();
    MoDevice.MoColumnSummaries.SyncAll();

    Rescue();
}

std::tuple<dim3, dim3> GPU::Core::GetGridBlockConfig(const xint FnDbgVal) const
{
    const auto LnColumnCount = (FnDbgVal) ? FnDbgVal : GetColumnCount();
    auto LnDown6 = RA::Pow(LnColumnCount, 1.0 / 6.0);
    while (RA::Pow(LnDown6, 6.0) < LnColumnCount)
        LnDown6++;

    const auto LnTarget = RA::Pow(LnDown6, 3.0);
    auto LnDown3 = RA::Pow(LnTarget, 1.0 / 3.0);
    while (RA::Pow(LnDown3, 3.0) < LnTarget)
        LnDown3++;

    auto LnGrid = dim3(LnDown6, LnDown6, LnDown6);

    auto LnX = LnDown3;
    auto LnY = LnDown3;
    auto LnZ = LnDown3;

    // find min block
    while ((pow(LnDown6, 3) * LnX * LnY * LnZ) > LnColumnCount)
    {
        if (LnY >= LnZ)
            LnZ--;
        else if (LnX >= LnY)
            LnY--;
        else
            LnX--;
    }
    // increase to hold enough values
    while (
        ((pow(LnDown6, 3) * LnX * LnY * LnZ) < LnColumnCount) 
        || ((LnX * LnY * LnZ) % 32 != 0))
    {
        if (LnY >= LnX)
            LnX++;
        else if (LnY >= LnZ)
            LnY++;
        else
            LnZ++;
    }
    auto LnBlock = dim3(LnX, LnY, LnZ);

    LnX = LnDown3;
    LnY = LnDown3;
    LnZ = LnDown3;
    while ((LnBlock.x * LnBlock.y * LnBlock.z * LnX * LnY * LnZ) > LnColumnCount)
    {
        if (LnY >= LnZ)
            LnZ--;
        else if (LnX >= LnY)
            LnY--;
        else
            LnX--;
    }
    while (
        ((LnBlock.x * LnBlock.y * LnBlock.z * LnX * LnY * LnZ) < LnColumnCount)
        /*|| ((LnX * LnY * LnZ) % 32 != 0)*/)
    {
        if (LnY >= LnX)
            LnX++;
        else if (LnY >= LnZ)
            LnY++;
        else
            LnZ++;
    }
    LnGrid.x = LnX;
    LnGrid.y = LnY;
    LnGrid.z = LnZ;


    RA::Print("Grid/Block: ",
        "(", LnGrid.x, ',', LnGrid.y, ',', LnGrid.z, ")",
        "(", LnBlock.x, ',', LnBlock.y, ',', LnBlock.z, ")");

    return std::make_tuple(LnGrid, LnBlock);
}

CST ColumnSummary& GPU::Core::GetDataset(const xint FnValue) CST
{ 
    Begin();
    return MoDevice.MoColumnSummaries[FnValue]; 
    Rescue();
}