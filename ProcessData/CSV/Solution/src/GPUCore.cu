#include "hip/hip_runtime.h"
#include "GPUCore.cuh"
#include "Timer.h"
#include "OS.h"
#include "CudaBridge.cuh"

__global__ void GPU::ParseResultColumnIdx(
    ColumnSummary* FvSummaries, 
    RA::StatsGPU* FvStats, const ColumnData* FvColumnData, 
    const xint FnColumnCount, const xint FnRowCount)
{
    auto Col = RA::Device::GetThreadID();
    if (Col >= FnColumnCount)
        return;
    const auto& LvColumnValues = FvColumnData[Col];
    auto& LoStats = FvStats[Col];
    auto& LoSummary = FvSummaries[Col];

    for (xint i = 0; i < FnRowCount; i++)
        LoStats << LvColumnValues.MvDeviceRows[i];
    
    LoSummary.Set(FnRowCount, LoStats);
}

GPU::Core::Core(const xstring& FsFilePath): APU::Core(FsFilePath)
{
}

GPU::Core::~Core()
{
    HostDelete(MoHost.MvColumnData);
}

void GPU::Core::ConfigureColumnValues()
{
    Begin();
    APU::Core::ConfigureColumnValues();

    MoHost.MvColumnData = new ColumnData[GetColumnCount()];
    for (xint Col = 0; Col < GetColumnCount(); Col++)
        MoHost.MvColumnData[Col].Initialize(MoHost.MvColumnValues[Col]);

    MoDevice.MvColumnData = 
        RA::Host::AllocateArrOnDevice<ColumnData>(
        MoHost.MvColumnData, RA::Allocate(GetColumnCount(), 
            sizeof(ColumnData)));

    Rescue();
}

void GPU::Core::ParseResults(const bool FbForceRestart)
{
    Begin();
    if(!FbForceRestart)
        if (MbParsed && MoDevice.MoResultStats.Size())
            return;

    const auto LnRetAllocate = RA::Allocate(GetColumnCount(), sizeof(ColumnData));
    const auto LmStatOps = xmap<RA::EStatOpt, xint>{
        {RA::EStatOpt::AVG, 0},{RA::EStatOpt::STOCH, 0},{RA::EStatOpt::SD, 0}
    };

    MoHost.MvStatsGPU = MKP<RA::StatsGPU[]>(GetColumnCount());
    for (auto& LoStat : MoHost.MvStatsGPU)
        LoStat.Construct(0, LmStatOps);

    MoDevice.MoResultStats = RA::CudaBridge<RA::StatsGPU>(MoHost.MvStatsGPU, MoHost.MvStatsGPU.GetLength());
    MoDevice.MoResultStats.AllocateHost();
    MoDevice.MoResultStats.AllocateDevice();
    MoDevice.MoResultStats.CopyHostToDeviceAsync();
    MoDevice.MoResultStats.SyncStream();

    const auto [LnGrid, LnBlock] = RA::Host::GetDimensions3D(GetColumnCount());
    MoDevice.MoColumnSummaries = RA::CudaBridge<ColumnSummary>::ARRAY::RunGPU(
        RA::Allocate(GetColumnCount(), sizeof(ColumnSummary)),
        LnGrid, LnBlock,
        &GPU::ParseResultColumnIdx,
        MoDevice.MoResultStats.GetDevice(),
        MoDevice.MvColumnData, GetColumnCount(), GetRowCount()
    );
    RA::CudaBridge<ColumnSummary>::SyncAll();
    MoDevice.MoColumnSummaries.CopyDeviceToHost();
    MoDevice.MoColumnSummaries.SyncAll();

    Rescue();
}

CST ColumnSummary& GPU::Core::GetDataset(const xint FnValue) CST
{ 
    Begin();
    return MoDevice.MoColumnSummaries[FnValue]; 
    Rescue();
}