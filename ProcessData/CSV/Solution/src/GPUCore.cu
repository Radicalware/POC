#include "hip/hip_runtime.h"
#include "GPUCore.cuh"
#include "Timer.h"
#include "OS.h"
#include "CudaBridge.cuh"


__global__ void GPU::ParseForColumnSummary(
    ColumnSummary* FvSummaries, 
    RA::StatsGPU* FvStats, const ColumnData* FvColumnData, 
    const xint FnColumnCount, const xint FnRowCount,
    const xint FnReloop)
{
    auto Col = RA::Device::GetThreadID();
    if (Col >= FnColumnCount)
        return;
    
    const auto& LvColumnValues = FvColumnData[Col];
    auto& LoStats = FvStats[Col];

    for (xint l = 0; l < FnReloop; l++)
    {
        for (xint i = 0; i < FnRowCount; i++)
            LoStats << LvColumnValues.MvDeviceRows[i];
    }
    
    FvSummaries[Col].SetGPU(FnRowCount, LoStats);
}

GPU::Core::Core(const xstring& FsFilePath): APU::Core(FsFilePath)
{
}

GPU::Core::~Core()
{
    HostDelete(MoHost.MvColumnData);
}

void GPU::Core::ConfigureColumnValues()
{
    Begin();
    APU::Core::ConfigureColumnValues();

    MoHost.MvColumnData = new ColumnData[GetColumnCount()];
    for (xint Col = 0; Col < GetColumnCount(); Col++)
        MoHost.MvColumnData[Col].Initialize(MoHost.MvColumnValues[Col]);

    MoDevice.MvColumnData = 
        RA::Host::AllocateArrOnDevice<ColumnData>(
        MoHost.MvColumnData, RA::Allocate(GetColumnCount(), 
            sizeof(ColumnData)));


    MoHost.MvStatsGPU = MKP<RA::StatsGPU[]>(GetColumnCount());
    for (auto& LoStat : MoHost.MvStatsGPU)
        LoStat.Construct(0, SvStatArgs);

    Rescue();
}

void GPU::Core::ParseResults(const bool FbForceRestart)
{
    Begin();

    MoDevice.MoResultStats = RA::CudaBridge<RA::StatsGPU>(MoHost.MvStatsGPU, MoHost.MvStatsGPU.GetLength());
    MoDevice.MoResultStats.AllocateDevice();
    MoDevice.MoResultStats.CopyHostToDeviceAsync();
    MoDevice.MoResultStats.SyncStream();

    cout << "Column Count: " << RA::FormatNum(GetColumnCount()) << endl;
    const auto [LvGrid, LvBlock] = RA::Host::GetDimensions3D(GetColumnCount());
    
    SetCudaMaxMem(GPU::ParseForColumnSummary);
    RA::Host::PrintGridBlockDims(LvGrid, LvBlock);
    MoDevice.MoColumnSummaries = RA::CudaBridge<ColumnSummary>::ARRAY::RunGPU(
        RA::Allocate(GetColumnCount(), sizeof(ColumnSummary)),
        LvGrid, LvBlock,
        &GPU::ParseForColumnSummary,
        MoDevice.MoResultStats.GetDevice(),
        MoDevice.MvColumnData, GetColumnCount(), GetRowCount(),
        SnReloop
    );
    MoDevice.MoColumnSummaries.SyncStream();
    MoDevice.MoColumnSummaries.CopyDeviceToHost();
    MoDevice.MoColumnSummaries.SyncAll();

    MoHost.MvSummaries = MoDevice.MoColumnSummaries.GetShared();

    Rescue();
}
