#include "hip/hip_runtime.h"
#include "Core.cuh"
#include "Timer.h"
#include "OS.h"

std::ostream& operator<<(std::ostream& out, const ColumnData& FoData)
{
    out <<
        "Count: " << FoData.Count    << '\n' <<
        "Mean:  " << FoData.Mean     << '\n' <<
        "Sum :  " << FoData.Sum      << '\n' <<
        "SD:    " << FoData.SD       << '\n' <<
        "Vrc:   " << FoData.Variance << '\n' <<
        "Min:   " << FoData.Min      << '\n' <<
        "Max:   " << FoData.Max      << '\n';
    return out;
}

Core::Core(const xstring& FsFilePath): MsFilePath(FsFilePath)
{
}

void Core::ReadData()
{
    Begin();
    MbParsed = false;
    auto LoTimer = RA::Timer();
    MvColumnValuesStr = RA::OS::ReadFile(MsFilePath)
        .Split('\n')
#ifdef BxDebug
        .ForEachThreadSeq<xvector<xstring>>([](const xstring& Str) { return Str.Split(','); });
#else // BxDebug
        .ForEachThreadSeq<xvector<xstring>>([](const xstring& Str) { return Str.Split(','); });
#endif
    MnColumnCount = MvColumnValuesStr.At(0).Size();
    cout << "Time to read data MS: " << LoTimer.GetElapsedTimeMilliseconds() << endl;
    MvColumnValues.clear();
    Rescue();
}

void Core::ConfigureColumnValues()
{
    Begin();
    if (MbParsed)
        return;
    MbParsed = true;
    MvBlankRow.clear();
    for (xint i = 0; i < MnColumnCount; i++)
    {
        MvBlankRow << 0;
        MvRange << i;
    }

    const auto LnSize = MvColumnValues.Size();
    for (xint Col = 0; Col < MnColumnCount; Col++)
    {
        MvColumnValues << xvector<double>();
        for (xint Row = 1; Row < MvColumnValuesStr.Size(); Row++) // start at idx 1 because idx 0 is the row descriptors
            if(MvColumnValuesStr[Row].Size()) // row of size 0 occurse at the EOF
                MvColumnValues.Last() << MvColumnValuesStr[Row][Col].ToDouble();
    }
    Rescue();
}

void Core::ParseResultColumnIdx(const xint Col)
{
    Begin();
    auto& LoData = MvResultData[Col];
    const auto& LvValues = MvColumnValues[Col];

    //LoData.Count = LValues.Size();
    LoData.Count = 1;

    LoData.Min = LvValues[0];
    LoData.Max = LvValues[0];
    LoData.Sum = LvValues[0];
    for (xint i = 1; i < LvValues.Size(); i++)
    {
        LoData.Count++;
        const auto& LnValue = LvValues[i];
        const auto& LnLastValue = LvValues[i - 1];
        if (LoData.Max < LnValue)
            LoData.Max = LnValue;
        if (LoData.Min > LnValue)
            LoData.Min = LnValue;
        LoData.Sum += LnValue;

        LoData.Mean = ((LoData.Mean * LoData.Count) + LnValue) / (LoData.Count + 1);
        LoData.SumDeviation += std::pow((LnValue - LoData.Mean), 2);
        LoData.SD = std::sqrt(LoData.SumDeviation / static_cast<double>(LoData.Count));
    }

    if (LoData.Count != LvValues.Size())
        ThrowIt("Size Mismatch!!");

    LoData.Mean = LoData.Sum / static_cast<double>(LoData.Count);
    for (const auto& Val : LvValues)
        LoData.Variance += std::pow((Val - LoData.Mean), 2);
    LoData.Variance /= static_cast<double>(LoData.Count);
    Rescue();
}

void Core::ParseResultsWtihCPU()
{
    Begin();
    if (MbParsed && MvResultData.Size())
        return;
    MvResultData.clear();
    for (xint i = 0; i < MnColumnCount; i++)
        MvResultData << ColumnData();

    for (xint Col = 0; Col < MnColumnCount; Col++)
        ParseResultColumnIdx(Col);

    Rescue();
}

void Core::ParseThreadedResultsWtihCPU(const bool FbForceRestart)
{
    Begin();

    if(!FbForceRestart)
        if (MbParsed && MvResultData.Size())
            return;

    MvResultData.clear();
    for (xint i = 0; i < MnColumnCount; i++)
        MvResultData << ColumnData();

    MvRange.LoopAllUnseq([this](const xint Idx) { The.ParseResultColumnIdx(Idx); });

    Rescue();
}
