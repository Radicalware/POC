#include "hip/hip_runtime.h"
#include "CPUCore.cuh"
#include "Timer.h"
#include "OS.h"
#include "Stats.cuh"
#include "Nexus.h"
#include "SYS.h"

CPU::Core::Core(const xstring& FsFilePath, const bool FbMultiCPU) : 
    APU::Core(FsFilePath), MbMultiCPU(FbMultiCPU)
{
}


void CPU::Core::ParseIndex(const xint FnCol)
{
    Begin();

    const auto& LvValues = MoHost.MvColumnValues[FnCol];
    auto& LoData = MoHost.MvSummaries[FnCol];

    //LoData.Count = LValues.Size();
    LoData.Count = 1;

    LoData.Min = LvValues[0];
    LoData.Max = LvValues[0];
    LoData.Sum = LvValues[0];
    for (xint i = 1; i < LvValues.Size(); i++)
    {
        LoData.Count++;
        const auto& LnValue = LvValues[i];
        const auto& LnLastValue = LvValues[i - 1];
        if (LoData.Max < LnValue)
            LoData.Max = LnValue;
        if (LoData.Min > LnValue)
            LoData.Min = LnValue;
        LoData.Sum += LnValue;

        LoData.Mean = ((LoData.Mean * LoData.Count) + LnValue) / (LoData.Count + 1);
        const auto LnValMinusMean = LnValue - LoData.Mean;
        LoData.SumDeviation += (LnValMinusMean * LnValMinusMean);
        LoData.SD = std::sqrt(LoData.SumDeviation / static_cast<double>(LoData.Count));
    }

    if (LoData.Count != LvValues.Size())
        ThrowIt("Size Mismatch!!");

    LoData.Mean = LoData.Sum / static_cast<double>(LoData.Count);
    for (const auto& Val : LvValues)
    {
        const auto LnValMinusMean = Val - LoData.Mean;
        LoData.Variance += (LnValMinusMean * LnValMinusMean);
    }
    LoData.Variance /= static_cast<double>(LoData.Count);

    Rescue();
}

void CPU::Core::ParseIndex(const xint FnCol, RA::StatsCPU& FoStat)
{
    Begin();

    for (xint l = 0; l < SnReloop; l++)
    {
        for (xint Row = 0; Row < GetRowCount(); Row++)
            FoStat << MoHost.MvColumnValues[FnCol][Row];
    }
    Rescue();
}

void CPU::Core::ParseIndicies(const xint FnCol)
{
    Begin();
    ParseIndex(FnCol, MoHost.MvStatsCPU[FnCol]);
    MoHost.MvSummaries[FnCol].SetCPU(GetRowCount(), MoHost.MvStatsCPU[FnCol]);
    Rescue();
}

void CPU::Core::ParseResults(const bool FbForceRestart)
{
    Begin();

    if (!FbForceRestart)
        if (MbParsed && MoHost.MvSummaries.Size())
            return;


    const auto LmStatOps = xmap<RA::EStatOpt, xint>{ 
        {RA::EStatOpt::AVG, 0},{RA::EStatOpt::STOCH, 0},{RA::EStatOpt::SD, 0} 
    };
    MoHost.MvStatsCPU = MKP<RA::StatsCPU[]>(GetColumnCount()/*, 0, LmStatOps*/);

    MoHost.MvSummaries = MKP<ColumnSummary[]>(GetColumnCount());

//#ifdef BxDebug
//    MbMultiCPU = false;
//#endif // BxDebug

    for (auto& LoStat : MoHost.MvStatsCPU)
        LoStat.Construct(0, LmStatOps);

    if (MbMultiCPU)
    {
        if (CliArgs.Has('j'))
        {
            auto LvThreads = xvector<xp<std::jthread>>();
            for (xint Col = 0; Col < GetColumnCount(); Col++)
                LvThreads << MKP<std::jthread>(std::bind(&CPU::Core::ParseIndicies, std::ref(The), Col));
            LvThreads.EraseAll();
        }
        else
        {
            for (xint Col = 0; Col < GetColumnCount(); Col++)
                Nexus<>::AddTask(The, &CPU::Core::ParseIndicies, Col);
            Nexus<>::WaitAll();
        }
    }
    else
    {
        MoHost.MvStatsCPU.Proc([this, &LmStatOps](auto& LoStat) { LoStat.Construct(0, LmStatOps); });
        for (xint Col = 0; Col < GetColumnCount(); Col++)
            ParseIndicies(Col);
    }

    Rescue();
}
