#include "hip/hip_runtime.h"
#include "CPUCore.cuh"
#include "Timer.h"
#include "OS.h"
#include "Stats.cuh"
#include "Nexus.h"
#include "SYS.h"

CPU::Core::Core(const xstring& FsFilePath, const bool FbMultiCPU) : 
    APU::Core(FsFilePath), MbMultiCPU(FbMultiCPU)
{
}


void CPU::Core::ParseIndex(const xint FnCol)
{
    Begin();

    const auto& LvValues = MoHost.MvColumnValues[FnCol];
    auto& LoData = MoHost.MvSummaries[FnCol];

    //LoData.Count = LValues.Size();
    LoData.Count = 1;

    LoData.Min = LvValues[0];
    LoData.Max = LvValues[0];
    LoData.Sum = LvValues[0];
    for (xint i = 1; i < LvValues.Size(); i++)
    {
        LoData.Count++;
        const auto& LnValue = LvValues[i];
        const auto& LnLastValue = LvValues[i - 1];
        if (LoData.Max < LnValue)
            LoData.Max = LnValue;
        if (LoData.Min > LnValue)
            LoData.Min = LnValue;
        LoData.Sum += LnValue;

        LoData.Mean = ((LoData.Mean * LoData.Count) + LnValue) / (LoData.Count + 1);
        const auto LnValMinusMean = LnValue - LoData.Mean;
        LoData.SumDeviation += (LnValMinusMean * LnValMinusMean);
        LoData.SD = std::sqrt(LoData.SumDeviation / static_cast<double>(LoData.Count));
    }

    if (LoData.Count != LvValues.Size())
        ThrowIt("Size Mismatch!!");

    LoData.Mean = LoData.Sum / static_cast<double>(LoData.Count);
    for (const auto& Val : LvValues)
    {
        const auto LnValMinusMean = Val - LoData.Mean;
        LoData.Variance += (LnValMinusMean * LnValMinusMean);
    }
    LoData.Variance /= static_cast<double>(LoData.Count);

    Rescue();
}

void CPU::Core::ParseForColumnSummary(const xint FnCol)
{
    Begin();
    auto& LoStats = MoHost.MvStatsCPU[FnCol];

    for (xint l = 0; l < SnReloop; l++)
    {
        for (xint Row = 0; Row < GetRowCount(); Row++)
            LoStats << MoHost.MvColumnValues[FnCol][Row];
    }

    MoHost.MvSummaries[FnCol].SetCPU(GetRowCount(), LoStats);
    Rescue();
}

void CPU::Core::ConfigureColumnValues()
{
    APU::Core::ConfigureColumnValues();
    MoHost.MvStatsCPU  = MKP<RA::StatsCPU[]>(GetColumnCount());
    MoHost.MvSummaries = MKP<ColumnSummary[]>(GetColumnCount());
    for (auto& LoStat : MoHost.MvStatsCPU)
        LoStat.Construct(0, SvStatArgs);
}

void CPU::Core::ParseResults(const bool FbForceRestart)
{
    Begin();

//#ifdef BxDebug
//    MbMultiCPU = false;
//#endif // BxDebug

    if (MbMultiCPU)
    {
        if (CliArgs.Has('j'))
        {
            auto LvThreads = xvector<xp<std::jthread>>();
            for (xint Col = 0; Col < GetColumnCount(); Col++)
                LvThreads << MKP<std::jthread>(std::bind(&CPU::Core::ParseForColumnSummary, std::ref(The), Col));
            LvThreads.EraseAll();
        }
        else
        {
            for (xint Col = 0; Col < GetColumnCount(); Col++)
                Nexus<>::AddTask(The, &CPU::Core::ParseForColumnSummary, Col);
            Nexus<>::WaitAll();
        }
    }
    else
    {
        for (xint Col = 0; Col < GetColumnCount(); Col++)
            ParseForColumnSummary(Col);
    }

    Rescue();
}
