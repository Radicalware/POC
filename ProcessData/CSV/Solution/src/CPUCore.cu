#include "hip/hip_runtime.h"
#include "CPUCore.cuh"
#include "Timer.h"
#include "OS.h"
#include "Stats.cuh"
#include "Nexus.h"

CPU::Core::Core(const xstring& FsFilePath, const bool FbMultiCPU) : 
    APU::Core(FsFilePath), MbMultiCPU(FbMultiCPU)
{
}


void CPU::Core::ParseIndex(const xint FnCol)
{
    Begin();

    const auto& LvValues = MoHost.MvColumnValues[FnCol];
    auto& LoData = MoHost.MvSummaries[FnCol];

    //LoData.Count = LValues.Size();
    LoData.Count = 1;

    LoData.Min = LvValues[0];
    LoData.Max = LvValues[0];
    LoData.Sum = LvValues[0];
    for (xint i = 1; i < LvValues.Size(); i++)
    {
        LoData.Count++;
        const auto& LnValue = LvValues[i];
        const auto& LnLastValue = LvValues[i - 1];
        if (LoData.Max < LnValue)
            LoData.Max = LnValue;
        if (LoData.Min > LnValue)
            LoData.Min = LnValue;
        LoData.Sum += LnValue;

        LoData.Mean = ((LoData.Mean * LoData.Count) + LnValue) / (LoData.Count + 1);
        LoData.SumDeviation += std::pow((LnValue - LoData.Mean), 2);
        LoData.SD = std::sqrt(LoData.SumDeviation / static_cast<double>(LoData.Count));
    }

    if (LoData.Count != LvValues.Size())
        ThrowIt("Size Mismatch!!");

    LoData.Mean = LoData.Sum / static_cast<double>(LoData.Count);
    for (const auto& Val : LvValues)
        LoData.Variance += std::pow((Val - LoData.Mean), 2);
    LoData.Variance /= static_cast<double>(LoData.Count);

    Rescue();
}

void CPU::Core::ParseIndex(const xint FnCol, RA::StatsCPU& FoStat)
{
    Begin();
    for (xint Row = 0; Row < GetRowCount(); Row++)
        FoStat << MoHost.MvColumnValues[FnCol][Row];
    Rescue();
}

void CPU::Core::ParseResults(const bool FbForceRestart)
{
    Begin();

    if (!FbForceRestart)
        if (MbParsed && MoHost.MvSummaries.Size())
            return;

    MoHost.MvSummaries.clear();
    MoHost.MvSummaries.resize(GetColumnCount());

    const auto LmStatOps = xmap<RA::EStatOpt, xint>{ 
        {RA::EStatOpt::AVG, 0},{RA::EStatOpt::STOCH, 0},{RA::EStatOpt::SD, 0} 
    };
    const xint LnZero = 0;
    MoHost.MvStatsCPU = MKP<RA::StatsCPU[]>(GetColumnCount()/*, 0, LmStatOps*/);

    if (MbMultiCPU)
    {
        for (RA::StatsCPU& LoStat : MoHost.MvStatsCPU)
            Nexus<void>::AddTask(LoStat, &RA::StatsCPU::Construct, LnZero, LmStatOps, LnZero);

        MvRange.LoopAllUnseq(
            [this](const xint FnCol)
            { The.ParseIndex(FnCol, MoHost.MvStatsCPU[FnCol]); return false; }
        );

        MvRange.LoopAllUnseq([this](const xint FnCol)
            { The.MoHost.MvSummaries[FnCol].Set(GetRowCount(), MoHost.MvStatsCPU[FnCol]); }
        );
    }
    else
    {
        MoHost.MvStatsCPU.Proc([this, &LmStatOps](auto& LoStat) { LoStat.Construct(0, LmStatOps); });

        MvRange.Proc(
            [this](const xint FnCol)
            { The.ParseIndex(FnCol, MoHost.MvStatsCPU[FnCol]); return false; }
        );

        MvRange.Proc([this](const xint FnCol)
            { The.MoHost.MvSummaries[FnCol].Set(GetRowCount(), MoHost.MvStatsCPU[FnCol]); return false; }
        );
    }

    Rescue();
}


CST ColumnSummary& CPU::Core::GetDataset(const xint FnValue) CST
{
    Begin();
    return MoHost.MvSummaries[FnValue];
    Rescue();
}
