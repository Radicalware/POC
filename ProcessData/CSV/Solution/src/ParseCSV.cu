﻿// Copyright[2021][Joel Leagues aka Scourge] under the Apache V2 Licence

#include "Macros.h"
#include "xmap.h"

#include "StatsGPU.cuh"
#include "CudaBridge.cuh"
#include "OS.h"
#include "Timer.h"

#include "GPUCore.cuh"
#include "CPUCore.cuh"
#include "SYS.h"

#if BxDebug
#include "vld.h"
#endif

using std::cout;
using std::endl;

RA::SYS Args;

void ExitEarly(const char* Err)
{
    cout << Err << endl;
    EXIT();
}


class Test
{
public:
    istatic xstring SoPath;
public:

    istatic RA::Mutex SoMutex;
    istatic sp<APU::Core> SoCoreCPUPtr;
    istatic sp<APU::Core> SoCoreGPUPtr;

    istatic void Prep(const char* FsPlatform, APU::Core& FoCore);

    istatic void PrepCPU();
    istatic void PrepGPU();

    istatic void RunCPU();
    istatic void RunGPU();

    istatic void CheckValues();

    istatic xint GetTargetIndex();
    istatic void SetPath();
    istatic auto GetPath() { return SoPath; }
};

int main(int argc, char** argv)
{
    Begin();
    Nexus<>::Start();

    Args.AddAlias('p', "--Path");
    Args.AddAlias('i', "--Index");
    Args.AddAlias('c', "--TestCPU");
    Args.AddAlias('g', "--TestGPU");
    Args.AddAlias('m', "--Multiplier");
    Args.AddAlias('s', "--SingleCPU");

    Args.SetArgs(argc, argv);

    bool LbParseCPU = Args.Has('c');
    bool LbParseGPU = Args.Has('g');
    if (!LbParseCPU && !LbParseGPU)
    {
        LbParseCPU = true;
        LbParseGPU = true;
    }

#ifdef BxDebug
    //LbParseCPU = true;  // true, false
    //LbParseGPU = false; // true, false
#endif

    Test::SetPath();
    if (LbParseCPU)
#if BxDebug
        Test::PrepCPU();
#else
        Nexus<void>::AddTask(&Test::PrepCPU);
#endif
    if (LbParseGPU)
#if BxDebug
        Test::PrepGPU();
#else
        Nexus<void>::AddTask(&Test::PrepGPU);
#endif
    if (!LbParseCPU && !LbParseGPU)
        ExitEarly("No Selection");
    Nexus<void>::WaitAll();
    cout << "\n\n";

    if (LbParseCPU)
        Test::RunCPU();
    if (LbParseGPU)
        Test::RunGPU();

    if (LbParseCPU && LbParseGPU)
        Test::CheckValues();

    FinalRescue();
    Nexus<>::Stop();
    return 0;
}

void Test::SetPath()
{
    Begin();
    SoPath = xstring();
    if (Args.Has('p'))
        SoPath = Args.Key('p').First();
#if BxDebug
    else
        SoPath = "C:/Source/git/POC/ProcessData/CSV/Data.csv";
#endif

    if (!SoPath.Match(R"(^.*(\.csv)$)"))
        ExitEarly("File must be a csv type");

    if (!RA::OS::HasFile(SoPath))
        ExitEarly("Path not found");

    Rescue();
}

void Test::Prep(const char* FsPlatform, APU::Core& FoCore)
{
    auto LoTimer = RA::Timer();
    const auto LnMultiplierSize = (Args.Has('m') ? Args.Key('m').First().To64() : 1);
    FoCore.ReadData(LnMultiplierSize);
    {
        Test::SoMutex.Wait();
        auto LoLock = Test::SoMutex.CreateLock();
        cout << FsPlatform << "Read Data Sec: " << RA::FormatNum(LoTimer.GetElapsedTimeSeconds()) << endl;

    }

    LoTimer.Reset();
    FoCore.ConfigureColumnValues();
    {
        Test::SoMutex.Wait();
        auto LoLock = Test::SoMutex.CreateLock();
        cout << FsPlatform << "Config Data Sec: " << RA::FormatNum(LoTimer.GetElapsedTimeSeconds()) << endl;

    }
}

void Test::PrepCPU()
{
    const auto LbSingleCPU = Args.Has('s');
    const auto LbMultiCPU = !LbSingleCPU;
    SoCoreCPUPtr = MKP<CPU::Core>(SoPath, LbMultiCPU);
    GET(SoCoreCPU);
    Test::Prep("CPU", SoCoreCPU);
}

void Test::PrepGPU()
{
    SoCoreGPUPtr = MKP<GPU::Core>(SoPath);
    GET(SoCoreGPU);
    Test::Prep("GPU", SoCoreGPU);
}

xint Test::GetTargetIndex()
{
    xint LnIdx = 0;
    if (Args.Has('i'))
    {
        LnIdx = Args.Key('i').First().To64();
        cout << "Inspecting: " << LnIdx << endl;
    }
    return LnIdx;
}

void Test::RunCPU()
{
    Begin();
    cout << "Running: " << __CLASS__ << '\n';
    
    GET(SoCoreCPU);
    auto LoTimer = RA::Timer();
    SoCoreCPU.ParseResults();

    const auto LbSingleCPU = Args.Has('s');
    if (LbSingleCPU)
    {
        cout << "Time Single Thread CPU Sec: " << RA::FormatNum(LoTimer.GetElapsedTimeSeconds()) << endl;
        cout << "Time Single Thread CPU MS : " << RA::FormatNum(LoTimer.GetElapsedTimeMilliseconds()) << endl;
    }
    else
    {
        cout << "Time Multi Thread CPU Sec: " << RA::FormatNum(LoTimer.GetElapsedTimeSeconds()) << endl;
        cout << "Time Multi Thread CPU MS : " << RA::FormatNum(LoTimer.GetElapsedTimeMilliseconds()) << endl;
    }

    auto LnIdx = GetTargetIndex();
    cout << SoCoreCPU.GetDataset(LnIdx) << endl;

    Rescue();
}

void Test::RunGPU()
{
    Begin();
    cout << "Running: " << __CLASS__ << '\n';

    GET(SoCoreGPU);
    auto LoTimer = RA::Timer();
    SoCoreGPU.ParseResults();
    cout << "Time Multi Thread GPU Sec: " << RA::FormatNum(LoTimer.GetElapsedTimeSeconds()) << endl;
    cout << "Time Multi Thread GPU MS : " << RA::FormatNum(LoTimer.GetElapsedTimeMilliseconds()) << endl;

    auto LnIdx = GetTargetIndex();
    cout << SoCoreGPU.GetDataset(LnIdx) << endl;
    Rescue();
}

void Test::CheckValues()
{
    Begin();
    GET(SoCoreCPU);
    GET(SoCoreGPU);

    if (SoCoreCPU.GetColumnCount() != SoCoreGPU.GetColumnCount())
        ThrowIt("Bad Col Count: ", SoCoreCPU.GetColumnCount(), " != ", SoCoreGPU.GetColumnCount());
    if (SoCoreCPU.GetRowCount() != SoCoreGPU.GetRowCount())
        ThrowIt("Bad Col Count: ", SoCoreCPU.GetRowCount(), " != ", SoCoreGPU.GetRowCount());

    const auto LnColCount = SoCoreGPU.GetColumnCount();

    auto LbBadTest = false;
    for (xint Col = 0; Col < LnColCount; Col++)
    {
        if (SoCoreCPU.GetDataset(Col) != SoCoreGPU.GetDataset(Col))
        {
            LbBadTest = true;
            cout << "Bad Case Idx: " << Col << endl;
            cout << "CPU ------------------------" << endl;
            cout << SoCoreCPU.GetDataset(Col) << endl;
            cout << "GPU ------------------------" << endl;
            cout << SoCoreGPU.GetDataset(Col) << endl;
        }
    }

    if (!LbBadTest)
        cout << "\nNo Bad Checks!!\n\n";

    Rescue();
}
