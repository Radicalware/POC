﻿// Copyright[2021][Joel Leagues aka Scourge] under the Apache V2 Licence

#include "Macros.h"
#include "xmap.h"

#include "StatsGPU.cuh"
#include "CudaBridge.cuh"
#include "OS.h"
#include "Timer.h"

#include "Core.cuh"
#include "SYS.h"

// #include "vld.h"

using std::cout;
using std::endl;

RA::SYS Args;

void RetEarly(const char* Err)
{
    cout << Err << endl;
    EXIT();
}

int main(int argc, char** argv)
{
    Begin();
    Nexus<>::Start();

    Args.AddAlias('p', "--Path");
    Args.AddAlias('i', "--Index");
    Args.SetArgs(argc, argv);

    auto LsPath = xstring();
    if (Args.Has('p'))
        LsPath = Args.Key('p').First();
    else
        LsPath = "C:/Source/Study/CodingChallenges/Contracts/Amazon/Robotics/Data.csv";

    if (!LsPath.Match(R"(^.*(\.csv)$)"))
        RetEarly("File must be a csv type");

    if (!RA::OS::HasFile(LsPath))
        RetEarly("Path not found");

    auto LoCore = Core(LsPath);
    LoCore.ReadData();
    LoCore.ConfigureColumnValues();
    
    auto LoTime = RA::Timer();
    LoCore.ParseResultsWtihCPU();
    const auto LoSingleThreadData = LoCore.GetDataset(0);
    cout << "Time Single Thread: " << LoTime.GetElapsedTimeMilliseconds() << endl;
    LoTime.Reset();
    LoCore.ParseThreadedResultsWtihCPU(true);
    const auto LoMultiThreadData = LoCore.GetDataset(0);
    cout << "Time Multi  Thread: " << LoTime.GetElapsedTimeMilliseconds() << endl;

    xint LnIdx = 0;
    if (Args.Has('i'))
    {
        LnIdx = Args.Key('i').First().To64();
        cout << "Inspecting: " << LnIdx << endl;
    }
    cout << LoCore.GetDataset(LnIdx) << endl;

    cout << "------------------------" << endl;
    cout << LoSingleThreadData << endl;
    cout << "------------------------" << endl;
    cout << LoMultiThreadData << endl;
    cout << "------------------------" << endl;
    cout << LoCore.GetDataset(LnIdx) << endl;
    cout << "------------------------" << endl;

    FinalRescue();
    Nexus<>::Stop();
    return 0;
}

