﻿// Copyright[2021][Joel Leagues aka Scourge] under the Apache V2 Licence

#include "Macros.h"
#include "CudaBridge.cuh"
#include "xmap.h"

#include "StatsGPU.cuh"
#include "CudaBridge.cuh"
#include "OS.h"
#include "Timer.h"

#include "GPUCore.cuh"
#include "CPUCore.cuh"
#include "SYS.h"

#if BxDebug
#include "vld.h"
#endif

using std::cout;
using std::endl;

void ExitEarly(const char* Err)
{
    cout << Err << endl;
    EXIT();
}


class Test
{
public:
    istatic xstring SoPath;
public:

    class Timers
    {
    public:
        istatic double SnLoadGPUVals = 0;
        istatic double SnLoadCPUVals = 0;

        istatic double SnParseGPU = 0;
        istatic double SnParseCPU = 0;
    };

    istatic bool SbPrintedDim = false;
    istatic RA::Mutex SoMutex;
    istatic sp<APU::Core> SoCoreCPUPtr;
    istatic sp<APU::Core> SoCoreGPUPtr;

    istatic void Prep(const char* FsPlatform, APU::Core& FoCore);

    istatic void PrepCPU();
    istatic void PrepGPU();

    istatic void RunCPU();
    istatic void RunGPU();

    istatic void CheckValues();

    istatic xint GetTargetIndex();
    istatic void SetPath();
    istatic auto GetPath() { return SoPath; }
};

//  -p .\Data.csv -m 1000 -r 800

int main(int argc, char** argv)
{
    Begin();
    Nexus<>::Start();
    RA::CudaBridge<>::SyncAll();

    CliArgs.AddAlias('p', "--Path");
    CliArgs.AddAlias('i', "--Index");
    CliArgs.AddAlias('c', "--TestCPU");
    CliArgs.AddAlias('g', "--TestGPU");
    CliArgs.AddAlias('m', "--Multiplier");
    CliArgs.AddAlias('s', "--SingleCPU");
    CliArgs.AddAlias('r', "--RowLock");
    CliArgs.AddAlias('a', "--Assert");
    CliArgs.AddAlias('j', "--jThread");
    CliArgs.AddAlias('l', "--Loop");

    CliArgs.SetArgs(argc, argv);

    bool LbParseCPU = CliArgs.Has('c');
    bool LbParseGPU = CliArgs.Has('g');
    if (!LbParseCPU && !LbParseGPU)
    {
        LbParseCPU = true;
        LbParseGPU = true;
    }

    const bool LbTestBoth = (LbParseCPU && LbParseGPU);

#ifdef BxDebug
    //LbParseCPU = true;  // true, false
    //LbParseGPU = false; // true, false
#endif

    Test::SetPath();
    if (LbParseCPU)
        #if BxDebug
            Test::PrepCPU();
        #else
            Nexus<void>::AddTask(&Test::PrepCPU);
        #endif

    if (LbParseGPU)
        #if BxDebug
            Test::PrepGPU();
        #else
            Nexus<void>::AddTask(&Test::PrepGPU);
        #endif

    if (!LbParseCPU && !LbParseGPU)
        ExitEarly("No Selection");


    if (CliArgs.Has('l'))
        APU::Core::SnReloop = CliArgs.Key('l').First().To64();

    Nexus<void>::WaitAll();
    cout << "\n\n";

    if (LbTestBoth)
    {
        cout << "CPU/GPU Load Time: "
            << RA::FormatNum(Test::Timers::SnLoadCPUVals / Test::Timers::SnLoadGPUVals, 4) << endl;
        cout << "GPU/CPU Load Time: "
            << RA::FormatNum(Test::Timers::SnLoadGPUVals / Test::Timers::SnLoadCPUVals, 4) << endl;
        cout << "\n\n";
    }

    if (LbParseCPU)
        Test::RunCPU();
    if (LbParseGPU)
        Test::RunGPU();

    if (LbTestBoth)
    {
        cout << "CPU/GPU  Run  Time: "
            << RA::FormatNum(Test::Timers::SnParseCPU / Test::Timers::SnParseGPU, 4) << endl;
        cout << "CPU/GPU Total Time: "
            << RA::FormatNum((Test::Timers::SnLoadCPUVals + Test::Timers::SnParseCPU) 
                / (Test::Timers::SnLoadGPUVals + Test::Timers::SnParseGPU), 4) << endl;
        cout << "\n\n";
    }

    auto LbCheckVals = CliArgs.Has('a');
    #if BxDebug
        LbCheckVals = true;
    #endif
    if (LbCheckVals && LbTestBoth)
        Test::CheckValues();

    FinalRescue();
    Nexus<>::Stop();
    return 0;
}

void Test::SetPath()
{
    Begin();
    SoPath = xstring();
    if (CliArgs.Has('p'))
        SoPath = CliArgs.Key('p').First();
    #if BxDebug
    else
        SoPath = "C:/Source/git/POC/ProcessData/CSV/Data.csv";
    #endif

    if (!SoPath.Match(R"(^.*(\.csv)$)"))
        ExitEarly("File must be a csv type");

    if (!RA::OS::HasFile(SoPath))
        ExitEarly("Path not found");

    Rescue();
}

void Test::Prep(const char* FsPlatform, APU::Core& FoCore)
{
    auto LoTimer = RA::Timer();
    const auto LnMultiplierSize = (CliArgs.Has('m') ? CliArgs.Key('m').First().To64() : 1);
    xstring LsColumnRowFormat = FoCore.ReadData(LnMultiplierSize);
    {
        Test::SoMutex.Wait();
        auto LoLock = Test::SoMutex.CreateLock();
        if(!SbPrintedDim)
            LsColumnRowFormat.Print();
        SbPrintedDim = true;
        cout << FsPlatform << "Read Data MS: " << RA::FormatNum(LoTimer.GetElapsedTimeMilliseconds()) << endl;

    }

    LoTimer.Reset();
    FoCore.ConfigureColumnValues();
    {
        Test::SoMutex.Wait();
        auto LoLock = Test::SoMutex.CreateLock();
        cout << FsPlatform << "Config Data MS: " << RA::FormatNum(LoTimer.GetElapsedTimeMilliseconds()) << endl;

    }
}

void Test::PrepCPU()
{
    const auto LbSingleCPU = CliArgs.Has('s');
    const auto LbMultiCPU = !LbSingleCPU;
    SoCoreCPUPtr = MKP<CPU::Core>(SoPath, LbMultiCPU);
    GET(SoCoreCPU);
    auto LoTimer = RA::Timer();
    Test::Prep("CPU", SoCoreCPU);
    Timers::SnLoadCPUVals = LoTimer.GetElapsedTimeMilliseconds();
}

void Test::PrepGPU()
{
    SoCoreGPUPtr = MKP<GPU::Core>(SoPath);
    GET(SoCoreGPU);
    auto LoTimer = RA::Timer();
    Test::Prep("GPU", SoCoreGPU);
    RA::CudaBridge<>::SyncAll();
    Timers::SnLoadGPUVals = LoTimer.GetElapsedTimeMilliseconds();
}

xint Test::GetTargetIndex()
{
    xint LnIdx = 0;
    if (CliArgs.Has('i'))
    {
        LnIdx = CliArgs.Key('i').First().To64();
        cout << "Inspecting: " << LnIdx << endl;
    }
    return LnIdx;
}

void Test::RunCPU()
{
    Begin();
    cout << "Running: " << __CLASS__ << '\n';
    
    GET(SoCoreCPU);
    auto LoTimer = RA::Timer();
    SoCoreCPU.ParseResults();
    Timers::SnParseCPU = LoTimer.GetElapsedTimeMicroseconds();
    const auto LbSingleCPU = CliArgs.Has('s');
    if (LbSingleCPU)
        cout << "Time Single Thread CPU MS : " << RA::FormatNum(LoTimer.GetElapsedTimeMilliseconds()) << endl;
    else
        cout << "Time Multi Thread CPU MS : " << RA::FormatNum(LoTimer.GetElapsedTimeMilliseconds()) << endl;

    auto LnIdx = GetTargetIndex();
    cout << SoCoreCPU.GetColumnSummary(LnIdx) << endl;

    Rescue();
}

void Test::RunGPU()
{
    Begin();
    cout << "Running: " << __CLASS__ << '\n';

    GET(SoCoreGPU);
    auto LoTimer = RA::Timer();
    SoCoreGPU.ParseResults();
    Timers::SnParseGPU = LoTimer.GetElapsedTimeMicroseconds();
    cout << "Time Multi Thread GPU MS : " << RA::FormatNum(LoTimer.GetElapsedTimeMilliseconds()) << endl;

    auto LnIdx = GetTargetIndex();
    cout << SoCoreGPU.GetColumnSummary(LnIdx) << endl;
    Rescue();
}

void Test::CheckValues()
{
    Begin();
    GET(SoCoreCPU);
    GET(SoCoreGPU);

    if (SoCoreCPU.GetColumnCount() != SoCoreGPU.GetColumnCount())
        ThrowIt("Bad Col Count: ", SoCoreCPU.GetColumnCount(), " != ", SoCoreGPU.GetColumnCount());
    if (SoCoreCPU.GetRowCount() != SoCoreGPU.GetRowCount())
        ThrowIt("Bad Col Count: ", SoCoreCPU.GetRowCount(), " != ", SoCoreGPU.GetRowCount());

    const auto LnColCount = SoCoreGPU.GetColumnCount();

    auto LbBadTest = false;
    for (xint Col = 0; Col < LnColCount; Col++)
    {
        if (SoCoreCPU.GetColumnSummary(Col) != SoCoreGPU.GetColumnSummary(Col))
        {
            LbBadTest = true;
            cout << "Bad Case Idx: " << Col << endl;
            cout << "CPU ------------------------" << endl;
            cout << SoCoreCPU.GetColumnSummary(Col) << endl;
            cout << "GPU ------------------------" << endl;
            cout << SoCoreGPU.GetColumnSummary(Col) << endl;
        }
    }

    if (!LbBadTest)
        cout << "No Bad Checks!!\n\n";

    Rescue();
}
