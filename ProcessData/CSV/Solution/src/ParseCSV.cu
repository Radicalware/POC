﻿// Copyright[2021][Joel Leagues aka Scourge] under the Apache V2 Licence

#include "Macros.h"
#include "xmap.h"

#include "StatsGPU.cuh"
#include "CudaBridge.cuh"
#include "OS.h"
#include "Timer.h"

#include "GPUCore.cuh"
#include "CPUCore.cuh"
#include "SYS.h"

#include "vld.h"

using std::cout;
using std::endl;

RA::SYS Args;

void RetEarly(const char* Err)
{
    cout << Err << endl;
    EXIT();
}

void TestCPU();
void TestGPU();
xstring GetPath();

int main(int argc, char** argv)
{
    Begin();
    Nexus<>::Start();

    Args.AddAlias('p', "--Path");
    Args.AddAlias('i', "--Index");
    Args.AddAlias('c', "--TestCPU");
    Args.AddAlias('g', "--TestGPU");
    Args.AddAlias('m', "--Multiplier");
    Args.AddAlias('s', "--SingleCPU");

    Args.SetArgs(argc, argv);

    bool LbParseCPU = Args.Has('c');
    bool LbParseGPU = Args.Has('g');
    if (!LbParseCPU && !LbParseGPU)
    {
        LbParseCPU = true;
        LbParseGPU = true;
    }

    if (LbParseCPU)
        TestCPU();
    if (LbParseGPU)
        TestGPU();

    FinalRescue();
    Nexus<>::Stop();
    return 0;
}

xstring GetPath()
{
    Begin();
    auto LsPath = xstring();
    if (Args.Has('p'))
        LsPath = Args.Key('p').First();
#if BxDebug
    else
        LsPath = "C:/Source/git/POC/ProcessData/CSV/Data.csv";
#endif

    if (!LsPath.Match(R"(^.*(\.csv)$)"))
        RetEarly("File must be a csv type");

    if (!RA::OS::HasFile(LsPath))
        RetEarly("Path not found");

    return LsPath;
    Rescue();
}

void TestAlgo(RA::Timer& FoTimer, APU::Core& FoCore)
{
    const auto LnMultiplierSize = (Args.Has('m') ? Args.Key('m').First().To64() : 1);
    FoCore.ReadData(LnMultiplierSize);
    FoCore.ConfigureColumnValues();

    FoTimer.Reset();
    FoCore.ParseResults();

    Rescue();
}

xint GetTargetIndex()
{
    xint LnIdx = 0;
    if (Args.Has('i'))
    {
        LnIdx = Args.Key('i').First().To64();
        cout << "Inspecting: " << LnIdx << endl;
    }
    return LnIdx;
}

void TestCPU()
{
    Begin();
    cout << "Running: " << __CLASS__ << '\n';

    const auto LbSingleGPU = Args.Has('s');
    const auto LbMultiGPU = !LbSingleGPU;
    xp<APU::Core> LoCorePtr = MKP<CPU::Core>(GetPath(), LbMultiGPU);
    GET(LoCore);
    auto LoTimer = RA::Timer();
    TestAlgo(LoTimer, LoCore);

    if(LbMultiGPU)
        cout << "Time Multi Thread CPU: " << LoTimer.GetElapsedTimeMilliseconds() << endl;
    else
        cout << "Time Single Thread CPU: " << LoTimer.GetElapsedTimeMilliseconds() << endl;


    auto LnIdx = GetTargetIndex();
    cout << LoCore.GetDataset(LnIdx) << endl;

    Rescue();
}

void TestGPU()
{
    Begin();
    cout << "Running: " << __CLASS__ << '\n';

    xp<APU::Core> LoCorePtr = MKP<GPU::Core>(GetPath());
    GET(LoCore);
    auto LoTimer = RA::Timer();
    TestAlgo(LoTimer, LoCore);
    cout << "Time Multi Thread GPU: " << LoTimer.GetElapsedTimeMilliseconds() << endl;

    auto LnIdx = GetTargetIndex();
    cout << LoCore.GetDataset(LnIdx) << endl;
    Rescue();
}